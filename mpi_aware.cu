#include <mpi.h>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

int main(int argc, char* argv[]) {
  	MPI_Init(&argc,&argv);
	{	
	int size;

	MPI_Comm_size(MPI_COMM_WORLD, &size);

	int rank;
	MPI_Comm_rank(MPI_COMM_WORLD, &rank);
	
	MPI_Status stat;

	if(size != 2){
		if(rank == 0){
			printf("This program requires exactly 2 MPI ranks, but you are attempting to use %d! Exiting...\n", size);
		}
		MPI_Finalize();
		exit(0);
	}
	
	//map MPI ranks to GPU
	hipSetDevice(rank);
	int tag1 = 10;
	int tag2 = 20;
	
	
for(int i = 0; i <= 29; i++) {
	long int N = 1 << i;	
	
	//device memoy
	double *device_buffer;
	hipMalloc(&device_buffer, N*sizeof(double));
	
	int loop_count = 50;
	
	auto start_time = MPI_Wtime();
	//warm-up loop
	//we have the staged version here
	for(int i = 1; i <=loop_count; i++) {
		if(rank == 0) {
			MPI_Send(device_buffer, N, MPI_DOUBLE, 1, tag1, MPI_COMM_WORLD);
			MPI_Recv(device_buffer, N, MPI_DOUBLE, 1, tag2, MPI_COMM_WORLD, &stat);

		}
		else if(rank == 1) {
			MPI_Recv(device_buffer, N, MPI_DOUBLE, 0, tag1, MPI_COMM_WORLD, &stat);
			MPI_Send(device_buffer, N, MPI_DOUBLE, 0, tag2, MPI_COMM_WORLD);
		}
	}

	auto stop_time = MPI_Wtime();
	auto elapsed_time = stop_time - start_time;

	//free the memory
	hipFree(device_buffer);

	long int num_B = 8*N;
	long int B_in_GB = 1 << 30;
	double num_GB = (double)num_B / (double)B_in_GB;
	double avg_time_per_transfer = elapsed_time / (2.0*(double)loop_count);

	if(rank == 0) {
		printf("Transfer size (B): %10li, Transfer Time (s): %15.9f, Bandwidth (GB/s): %15.9f\n", num_B, avg_time_per_transfer, num_GB/avg_time_per_transfer );
	}
}
	}
	MPI_Finalize();
}
